#include "hip/hip_runtime.h"
#include "Particles.h"
#include "Alloc.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "GPUAllocation.h"
#include <stdio.h>

#define STREAMS 2  // number of streams for each computation on the GPU

/** allocate particle arrays */
void particle_allocate(struct parameters* param, struct particles* part, int is)
{
    
    // set species ID
    part->species_ID = is;
    // number of particles
    part->nop = param->np[is];
    // maximum number of particles
    part->npmax = param->npMax[is];
    
    // choose a different number of mover iterations for ions and electrons
    if (param->qom[is] < 0){  //electrons
        part->NiterMover = param->NiterMover;
        part->n_sub_cycles = param->n_sub_cycles;
    } else {                  // ions: only one iteration
        part->NiterMover = 1;
        part->n_sub_cycles = 1;
    }
    
    // particles per cell
    part->npcelx = param->npcelx[is];
    part->npcely = param->npcely[is];
    part->npcelz = param->npcelz[is];
    part->npcel = part->npcelx*part->npcely*part->npcelz;
    
    // cast it to required precision
    part->qom = (FPpart) param->qom[is];
    
    long npmax = part->npmax;
    
    // initialize drift and thermal velocities
    // drift
    part->u0 = (FPpart) param->u0[is];
    part->v0 = (FPpart) param->v0[is];
    part->w0 = (FPpart) param->w0[is];
    // thermal
    part->uth = (FPpart) param->uth[is];
    part->vth = (FPpart) param->vth[is];
    part->wth = (FPpart) param->wth[is];
    
    
    //////////////////////////////
    /// ALLOCATION PARTICLE ARRAYS
    //////////////////////////////
    part->x = new FPpart[npmax];
    part->y = new FPpart[npmax];
    part->z = new FPpart[npmax];
    // allocate velocity
    part->u = new FPpart[npmax];
    part->v = new FPpart[npmax];
    part->w = new FPpart[npmax];
    // allocate charge = q * statistical weight
    part->q = new FPinterp[npmax];
    
}
/** deallocate */
void particle_deallocate(struct particles* part)
{
    // deallocate particle variables
    delete[] part->x;
    delete[] part->y;
    delete[] part->z;
    delete[] part->u;
    delete[] part->v;
    delete[] part->w;
    delete[] part->q;
}

/** deallocate (pinned memory)*/
void particle_deallocate_pinned(struct particles* part)
{
    // deallocate particle variables
    hipHostFree(part->x);
    hipHostFree(part->y);
    hipHostFree(part->z);
    hipHostFree(part->u);
    hipHostFree(part->v);
    hipHostFree(part->w);
    hipHostFree(part->q);
}

/** Compute number of batches */
int get_nob(int nop, int batchsize)
{
    return (nop + batchsize - 1) / batchsize;
}

/** Create batches of particles */
int particle_batch_create(struct parameters* param, struct particles* part, struct particles** part_batches)
{
    // Compute number of batches
    int nob = get_nob(part->nop, param->batchsize);

    // Fill one batch at a time with particle data
    // *part_batches = new particles[nob];
    hipHostMalloc((void **) part_batches, sizeof(particles)*nob, hipHostMallocDefault);

    for (int batch_id=0; batch_id<nob; ++batch_id) {
        // copy structure, pointers will still point to the same memory address
        (*part_batches)[batch_id] = *part;

        //////////////////////////////////////
        // Overwrite relevant scalar values //
        //////////////////////////////////////

        // number of particles
        if (batch_id == nob-1) {
            // Last batch is a special case as it may contain fewer than batchsize particles
            int batch_remainder = part->nop % param->batchsize;
            if (batch_remainder == 0)
                (*part_batches)[batch_id].nop = param->batchsize;
            else
                (*part_batches)[batch_id].nop = batch_remainder;
        }
        else
            (*part_batches)[batch_id].nop = param->batchsize;

        // maximum number of particles
        long npmax = param->batchsize; // I am not really sure if we can just use the batchsize here... 
        (*part_batches)[batch_id].npmax = npmax;
            
        ///////////////////////
        /// Overwrite arrays //
        ///////////////////////

        // Allocate new memory addresses
        // (*part_batches)[batch_id].x = new FPpart[npmax];
        // (*part_batches)[batch_id].y = new FPpart[npmax];
        // (*part_batches)[batch_id].z = new FPpart[npmax];
        // (*part_batches)[batch_id].u = new FPpart[npmax];
        // (*part_batches)[batch_id].v = new FPpart[npmax];
        // (*part_batches)[batch_id].w = new FPpart[npmax];
        // (*part_batches)[batch_id].q = new FPinterp[npmax];
        hipHostMalloc((void **) &((*part_batches)[batch_id].x), sizeof(FPpart)*npmax, hipHostMallocDefault);
        hipHostMalloc((void **) &((*part_batches)[batch_id].y), sizeof(FPpart)*npmax, hipHostMallocDefault);
        hipHostMalloc((void **) &((*part_batches)[batch_id].z), sizeof(FPpart)*npmax, hipHostMallocDefault);
        hipHostMalloc((void **) &((*part_batches)[batch_id].u), sizeof(FPpart)*npmax, hipHostMallocDefault);
        hipHostMalloc((void **) &((*part_batches)[batch_id].v), sizeof(FPpart)*npmax, hipHostMallocDefault);
        hipHostMalloc((void **) &((*part_batches)[batch_id].w), sizeof(FPpart)*npmax, hipHostMallocDefault);
        hipHostMalloc((void **) &((*part_batches)[batch_id].q), sizeof(FPpart)*npmax, hipHostMallocDefault);

        // Copy the values
        std::copy((part->x)+batch_id*param->batchsize, (part->x)+batch_id*param->batchsize+(*part_batches)[batch_id].nop, (*part_batches)[batch_id].x);
        std::copy((part->y)+batch_id*param->batchsize, (part->y)+batch_id*param->batchsize+(*part_batches)[batch_id].nop, (*part_batches)[batch_id].y);
        std::copy((part->z)+batch_id*param->batchsize, (part->z)+batch_id*param->batchsize+(*part_batches)[batch_id].nop, (*part_batches)[batch_id].z);
        std::copy((part->u)+batch_id*param->batchsize, (part->u)+batch_id*param->batchsize+(*part_batches)[batch_id].nop, (*part_batches)[batch_id].u);
        std::copy((part->v)+batch_id*param->batchsize, (part->v)+batch_id*param->batchsize+(*part_batches)[batch_id].nop, (*part_batches)[batch_id].v);
        std::copy((part->w)+batch_id*param->batchsize, (part->w)+batch_id*param->batchsize+(*part_batches)[batch_id].nop, (*part_batches)[batch_id].w);
        std::copy((part->q)+batch_id*param->batchsize, (part->q)+batch_id*param->batchsize+(*part_batches)[batch_id].nop, (*part_batches)[batch_id].q);
    }

    return nob;
}

/** Deallocate particle batches */
void particle_batch_deallocate(struct particles* part_batches, int nob)
{
    for (int i=0; i<nob; ++i)
    {
        particle_deallocate_pinned(&part_batches[i]);
    }
}

/** particle mover */
__global__
void mover_PC_gpu(struct particles* part, struct EMfield* field, struct grid* grd, struct parameters* param, long offset, long num_elem)
{
    // get thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // add offset to get global particle ID
    id = offset + id;
        
    // auxiliary variables
    FPpart dt_sub_cycling = (FPpart) param->dt/((double) part->n_sub_cycles);
    FPpart dto2 = .5*dt_sub_cycling, qomdt2 = part->qom*dto2/param->c;
    FPpart omdtsq, denom, ut, vt, wt, udotb;
    
    // local (to the particle) electric and magnetic field
    FPfield Exl=0.0, Eyl=0.0, Ezl=0.0, Bxl=0.0, Byl=0.0, Bzl=0.0;
    
    // interpolation densities
    int ix,iy,iz;
    FPfield weight[2][2][2];
    FPfield xi[2], eta[2], zeta[2];
    
    // intermediate particle position and velocity
    FPpart xptilde, yptilde, zptilde, uptilde, vptilde, wptilde;
    
    // start subcycling
    for (int i_sub=0; i_sub <  part->n_sub_cycles; i_sub++){
        // move each particle with new fields
        if (id < offset + num_elem){
            xptilde = part->x[id];
            yptilde = part->y[id];
            zptilde = part->z[id];
            // calculate the average velocity iteratively
            for(int innter=0; innter < part->NiterMover; innter++){
                // interpolation G-->P
                ix = 2 +  int((part->x[id] - grd->xStart)*grd->invdx);
                iy = 2 +  int((part->y[id] - grd->yStart)*grd->invdy);
                iz = 2 +  int((part->z[id] - grd->zStart)*grd->invdz);
                
                // calculate weights
                // xi[0]   = part->x[id] - grd->XN[ix - 1][iy][iz];
                // eta[0]  = part->y[id] - grd->YN[ix][iy - 1][iz];
                // zeta[0] = part->z[id] - grd->ZN[ix][iy][iz - 1];
                xi[0]   = part->x[id] - grd->XN_flat[get_idx(ix - 1, iy, iz, grd->nyn, grd->nzn)];                
                eta[0]  = part->y[id] - grd->YN_flat[get_idx(ix, iy - 1, iz, grd->nyn, grd->nzn)];                
                zeta[0] = part->z[id] - grd->ZN_flat[get_idx(ix, iy, iz - 1, grd->nyn, grd->nzn)];
                // xi[1]   = grd->XN[ix][iy][iz] - part->x[id];
                // eta[1]  = grd->YN[ix][iy][iz] - part->y[id];
                // zeta[1] = grd->ZN[ix][iy][iz] - part->z[id];
                xi[1]   = grd->XN_flat[get_idx(ix, iy, iz, grd->nyn, grd->nzn)] - part->x[id];                
                eta[1]  = grd->YN_flat[get_idx(ix, iy, iz, grd->nyn, grd->nzn)] - part->y[id];                
                zeta[1] = grd->ZN_flat[get_idx(ix, iy, iz, grd->nyn, grd->nzn)] - part->z[id];
                for (int ii = 0; ii < 2; ii++)
                    for (int jj = 0; jj < 2; jj++)
                        for (int kk = 0; kk < 2; kk++)
                            weight[ii][jj][kk] = xi[ii] * eta[jj] * zeta[kk] * grd->invVOL;
                
                // set to zero local electric and magnetic field
                Exl=0.0, Eyl = 0.0, Ezl = 0.0, Bxl = 0.0, Byl = 0.0, Bzl = 0.0;
                
                for (int ii=0; ii < 2; ii++)
                    for (int jj=0; jj < 2; jj++)
                        for(int kk=0; kk < 2; kk++){
                            // Exl += weight[ii][jj][kk]*field->Ex[ix- ii][iy -jj][iz- kk ];
                            // Eyl += weight[ii][jj][kk]*field->Ey[ix- ii][iy -jj][iz- kk ];
                            // Ezl += weight[ii][jj][kk]*field->Ez[ix- ii][iy -jj][iz -kk ];
                            // Bxl += weight[ii][jj][kk]*field->Bxn[ix- ii][iy -jj][iz -kk ];
                            // Byl += weight[ii][jj][kk]*field->Byn[ix- ii][iy -jj][iz -kk ];
                            // Bzl += weight[ii][jj][kk]*field->Bzn[ix- ii][iy -jj][iz -kk ];
                            Exl += weight[ii][jj][kk]*field->Ex_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)];
                            Eyl += weight[ii][jj][kk]*field->Ey_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)];
                            Ezl += weight[ii][jj][kk]*field->Ez_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)];
                            Bxl += weight[ii][jj][kk]*field->Bxn_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)];
                            Byl += weight[ii][jj][kk]*field->Byn_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)];
                            Bzl += weight[ii][jj][kk]*field->Bzn_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)];
                        }
                
                // end interpolation
                omdtsq = qomdt2*qomdt2*(Bxl*Bxl+Byl*Byl+Bzl*Bzl);
                denom = 1.0/(1.0 + omdtsq);
                // solve the position equation
                ut= part->u[id] + qomdt2*Exl;
                vt= part->v[id] + qomdt2*Eyl;
                wt= part->w[id] + qomdt2*Ezl;
                udotb = ut*Bxl + vt*Byl + wt*Bzl;
                // solve the velocity equation
                uptilde = (ut+qomdt2*(vt*Bzl -wt*Byl + qomdt2*udotb*Bxl))*denom;
                vptilde = (vt+qomdt2*(wt*Bxl -ut*Bzl + qomdt2*udotb*Byl))*denom;
                wptilde = (wt+qomdt2*(ut*Byl -vt*Bxl + qomdt2*udotb*Bzl))*denom;
                // update position
                part->x[id] = xptilde + uptilde*dto2;
                part->y[id] = yptilde + vptilde*dto2;
                part->z[id] = zptilde + wptilde*dto2;
                
                
            } // end of iteration
            // update the final position and velocity
            part->u[id]= 2.0*uptilde - part->u[id];
            part->v[id]= 2.0*vptilde - part->v[id];
            part->w[id]= 2.0*wptilde - part->w[id];
            part->x[id] = xptilde + uptilde*dt_sub_cycling;
            part->y[id] = yptilde + vptilde*dt_sub_cycling;
            part->z[id] = zptilde + wptilde*dt_sub_cycling;
            
            
            //////////
            //////////
            ////////// BC
                                        
            // X-DIRECTION: BC particles
            if (part->x[id] > grd->Lx){
                if (param->PERIODICX==true){ // PERIODIC
                    part->x[id] = part->x[id] - grd->Lx;
                } else { // REFLECTING BC
                    part->u[id] = -part->u[id];
                    part->x[id] = 2*grd->Lx - part->x[id];
                }
            }
                                                                        
            if (part->x[id] < 0){
                if (param->PERIODICX==true){ // PERIODIC
                   part->x[id] = part->x[id] + grd->Lx;
                } else { // REFLECTING BC
                    part->u[id] = -part->u[id];
                    part->x[id] = -part->x[id];
                }
            }
                
            
            // Y-DIRECTION: BC particles
            if (part->y[id] > grd->Ly){
                if (param->PERIODICY==true){ // PERIODIC
                    part->y[id] = part->y[id] - grd->Ly;
                } else { // REFLECTING BC
                    part->v[id] = -part->v[id];
                    part->y[id] = 2*grd->Ly - part->y[id];
                }
            }
                                                                        
            if (part->y[id] < 0){
                if (param->PERIODICY==true){ // PERIODIC
                    part->y[id] = part->y[id] + grd->Ly;
                } else { // REFLECTING BC
                    part->v[id] = -part->v[id];
                    part->y[id] = -part->y[id];
                }
            }
                                                                        
            // Z-DIRECTION: BC particles
            if (part->z[id] > grd->Lz){
                if (param->PERIODICZ==true){ // PERIODIC
                    part->z[id] = part->z[id] - grd->Lz;
                } else { // REFLECTING BC
                    part->w[id] = -part->w[id];
                    part->z[id] = 2*grd->Lz - part->z[id];
                }
            }
                                                                        
            if (part->z[id] < 0){
                if (param->PERIODICZ==true){ // PERIODIC
                    part->z[id] = part->z[id] + grd->Lz;
                } else { // REFLECTING BC
                    part->w[id] = -part->w[id];
                    part->z[id] = -part->z[id];
                }
            }
                                                                        
            
            
        }  // end of subcycling
    } // end of one particle
                                                                        
    return; // exit
}

/* launch GPU version of the particle mover */
int mover_PC_gpu_launch(struct particles* part, struct EMfield* field, struct grid* grd, struct parameters* param)
{
    // Copy EMfield struct to device
    EMfield* field_gpu;
    emfield_move2gpu(field, &field_gpu, grd);

    // Copy grid struct to device
    grid* grd_gpu;
    grid_move2gpu(grd, &grd_gpu);
    
    // Copy parameters struct to device
    parameters* param_gpu;
    hipMalloc(&param_gpu, sizeof(parameters));
    hipMemcpy(param_gpu, param, sizeof(parameters), hipMemcpyHostToDevice);

    ///////////////////////////////////////////////////////////////////////////////////////////////////
    // Divide the particle data in segments and use streams to overlap data transfer and computation //
    ///////////////////////////////////////////////////////////////////////////////////////////////////
    // Prepare auxiliary variables
    long pps = ceil(part->npmax / STREAMS);  // particles per stream
    long stream_offset[STREAMS];             // array segment offset
    long np_stream[STREAMS];                 // number of particles in stream

    // Create cuda streams and offsets and assign a number of particles to each stream
    hipStream_t stream[STREAMS];
    for (int s_id=0; s_id<STREAMS; ++s_id)
    {
        hipStreamCreate(&stream[s_id]);
        
        // Compute offset to specify start of array segments
        stream_offset[s_id] = s_id * pps;

        // Number of particles in stream is either equal to pps or what is left in the last stream
        np_stream[s_id] = std::min(pps, part->nop - stream_offset[s_id]); 
    }

    // Trigger asynchronous copy for each stream
    particles* part_gpu;
    particle_move2gpu(part, &part_gpu, STREAMS, stream, stream_offset, np_stream);

    // Launch kernels for each stream
    for (int s_id=0; s_id<STREAMS; ++s_id)
    {   
        // Call kernel (the third execution configuration parameter is 0 because no shared device memory is allocated)
        mover_PC_gpu<<<(np_stream[s_id]+param->tpb-1)/param->tpb, param->tpb, 0, stream[s_id]>>>(part_gpu, field_gpu, grd_gpu, param_gpu, stream_offset[s_id], np_stream[s_id]);
    }

    // Retrieve data from the device (trigger asynchronous copy)
    particle_move2cpu(part_gpu, part, STREAMS, stream, stream_offset, np_stream);
    
    // wait for GPU operations to finish and destroy streams
    hipDeviceSynchronize();
    for (int s_id=0; s_id<STREAMS; ++s_id)
    {
        hipStreamDestroy(stream[s_id]);
    }

    // Free the memory
    particle_deallocate_gpu(part_gpu);
    emfield_deallocate_gpu(field_gpu);
    grid_deallocate_gpu(grd_gpu);
    hipFree(param_gpu);

    return 0;
}


/** Interpolation Particle --> Grid: This is for species */
__global__
void interpP2G_gpu(struct particles* part, struct interpDensSpecies* ids, struct grid* grd, long offset, long num_elem)
{ 
    // get thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // add offset to get global particle ID
    id = offset + id;

    // arrays needed for interpolation
    FPpart weight[2][2][2];
    FPpart temp[2][2][2];
    FPpart xi[2], eta[2], zeta[2];
    
    // index of the cell
    int ix, iy, iz;
     
    if (id < offset + num_elem) {

        // determine cell: can we change to int()? is it faster?
        ix = 2 + int (floor((part->x[id] - grd->xStart) * grd->invdx));
        iy = 2 + int (floor((part->y[id] - grd->yStart) * grd->invdy));
        iz = 2 + int (floor((part->z[id] - grd->zStart) * grd->invdz));
        
        // distances from node
        // xi[0]   = part->x[id] - grd->XN[ix - 1][iy][iz];
        // eta[0]  = part->y[id] - grd->YN[ix][iy - 1][iz];
        // zeta[0] = part->z[id] - grd->ZN[ix][iy][iz - 1];
        xi[0]   = part->x[id] - grd->XN_flat[get_idx(ix - 1, iy, iz, grd->nyn, grd->nzn)];
        eta[0]  = part->y[id] - grd->YN_flat[get_idx(ix, iy - 1, iz, grd->nyn, grd->nzn)];
        zeta[0] = part->z[id] - grd->ZN_flat[get_idx(ix, iy, iz - 1, grd->nyn, grd->nzn)];
        // xi[1]   = grd->XN[ix][iy][iz] - part->x[id];
        // eta[1]  = grd->YN[ix][iy][iz] - part->y[id];
        // zeta[1] = grd->ZN[ix][iy][iz] - part->z[id];
        xi[1]   = grd->XN_flat[get_idx(ix, iy, iz, grd->nyn, grd->nzn)] - part->x[id];
        eta[1]  = grd->YN_flat[get_idx(ix, iy, iz, grd->nyn, grd->nzn)] - part->y[id];
        zeta[1] = grd->ZN_flat[get_idx(ix, iy, iz, grd->nyn, grd->nzn)] - part->z[id];
        
        // calculate the weights for different nodes
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    weight[ii][jj][kk] = part->q[id] * xi[ii] * eta[jj] * zeta[kk] * grd->invVOL;
        
        //////////////////////////
        // add charge density
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    // ids->rhon_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)] += weight[ii][jj][kk] * grd->invVOL;
                    atomicAdd(&(ids->rhon_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)]), weight[ii][jj][kk] * grd->invVOL);
        
        
        ////////////////////////////
        // add current density - Jx
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->u[id] * weight[ii][jj][kk];
        
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    // ids->Jx_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)] += temp[ii][jj][kk] * grd->invVOL;
                    atomicAdd(&(ids->Jx_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)]), temp[ii][jj][kk] * grd->invVOL);
        
        
        ////////////////////////////
        // add current density - Jy
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->v[id] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    // ids->Jy_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)] += temp[ii][jj][kk] * grd->invVOL;
                    atomicAdd(&(ids->Jy_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)]), temp[ii][jj][kk] * grd->invVOL);
        
        
        
        ////////////////////////////
        // add current density - Jz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->w[id] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    // ids->Jz_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)] += temp[ii][jj][kk] * grd->invVOL;
                    atomicAdd(&(ids->Jz_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)]), temp[ii][jj][kk] * grd->invVOL);
        
        
        ////////////////////////////
        // add pressure pxx
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->u[id] * part->u[id] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    // ids->pxx_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)] += temp[ii][jj][kk] * grd->invVOL;
                    atomicAdd(&(ids->pxx_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)]), temp[ii][jj][kk] * grd->invVOL);
        
        
        ////////////////////////////
        // add pressure pxy
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->u[id] * part->v[id] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    // ids->pxy_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)] += temp[ii][jj][kk] * grd->invVOL;
                    atomicAdd(&(ids->pxy_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)]), temp[ii][jj][kk] * grd->invVOL);
        
        
        
        /////////////////////////////
        // add pressure pxz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->u[id] * part->w[id] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    // ids->pxz_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)] += temp[ii][jj][kk] * grd->invVOL;
                    atomicAdd(&(ids->pxz_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)]), temp[ii][jj][kk] * grd->invVOL);
        
        
        /////////////////////////////
        // add pressure pyy
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->v[id] * part->v[id] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    // ids->pyy_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)] += temp[ii][jj][kk] * grd->invVOL;
                    atomicAdd(&(ids->pyy_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)]), temp[ii][jj][kk] * grd->invVOL);
        
        
        /////////////////////////////
        // add pressure pyz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->v[id] * part->w[id] * weight[ii][jj][kk];
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    // ids->pyz_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)] += temp[ii][jj][kk] * grd->invVOL;
                    atomicAdd(&(ids->pyz_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)]), temp[ii][jj][kk] * grd->invVOL);
        
        
        /////////////////////////////
        // add pressure pzz
        for (int ii = 0; ii < 2; ii++)
            for (int jj = 0; jj < 2; jj++)
                for (int kk = 0; kk < 2; kk++)
                    temp[ii][jj][kk] = part->w[id] * part->w[id] * weight[ii][jj][kk];
        for (int ii=0; ii < 2; ii++)
            for (int jj=0; jj < 2; jj++)
                for(int kk=0; kk < 2; kk++)
                    // ids->pzz_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)] += temp[ii][jj][kk] * grd->invVOL;
                    atomicAdd(&(ids->pzz_flat[get_idx(ix-ii, iy-jj, iz-kk, grd->nyn, grd->nzn)]), temp[ii][jj][kk] * grd->invVOL);
    
    }
   
}

/* launch GPU version of the P2G interpolation */
int interpP2G_gpu_launch(struct particles* part, struct interpDensSpecies* ids, struct grid* grd, struct parameters* param)
{
    // Copy interpDensSpecies struct to device
    interpDensSpecies* ids_gpu;
    ids_move2gpu(ids, &ids_gpu, grd);

    // Copy grid struct to device
    grid* grd_gpu;
    grid_move2gpu(grd, &grd_gpu);

    ///////////////////////////////////////////////////////////////////////////////////////////////////
    // Divide the particle data in segments and use streams to overlap data transfer and computation //
    ///////////////////////////////////////////////////////////////////////////////////////////////////
    // Prepare auxiliary variables
    long pps = ceil(part->npmax / STREAMS);  // particles per stream
    long stream_offset[STREAMS];             // array segment offset
    long np_stream[STREAMS];                 // number of particles in stream

    // Create cuda streams and offsets and assign a number of particles to each stream
    hipStream_t stream[STREAMS];
    for (int s_id=0; s_id<STREAMS; ++s_id)
    {
        hipStreamCreate(&stream[s_id]);
        
        // Compute offset to specify start of array segments
        stream_offset[s_id] = s_id * pps;

        // Number of particles in stream is either equal to pps or what is left in the last stream
        np_stream[s_id] = std::min(pps, part->nop - stream_offset[s_id]); 
    }

    // Trigger asynchronous copy for each stream
    particles* part_gpu;
    particle_move2gpu(part, &part_gpu, STREAMS, stream, stream_offset, np_stream);

    // Launch kernels for each stream
    for (int s_id=0; s_id<STREAMS; ++s_id)
    {
        // Call kernel (the third execution configuration parameter is 0 because no shared device memory is be allocated)
        interpP2G_gpu<<<(np_stream[s_id]+param->tpb-1)/param->tpb, param->tpb, 0, stream[s_id]>>>(part_gpu, ids_gpu, grd_gpu, stream_offset[s_id], np_stream[s_id]);
    }

    // wait for GPU operations to finish and destroy streams
    hipDeviceSynchronize();
    for (int s_id=0; s_id<STREAMS; ++s_id)
    {
        hipStreamDestroy(stream[s_id]);
    }

    // Retrieve data from the device
    ids_move2cpu(ids_gpu, ids, grd);

    // Free the memory
    particle_deallocate_gpu(part_gpu);
    ids_deallocate_gpu(ids_gpu);
    grid_deallocate_gpu(grd_gpu);

    return 0;
}

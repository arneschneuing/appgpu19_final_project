#include "GPUAllocation.h"

/** move particle array to GPU */
void particle_move2gpu(struct particles* part, struct particles** part_gpu)
{   
    // Allocate memory on the GPU
    hipMalloc(part_gpu, sizeof(particles)); 

    // Allocate arrays on the device
    FPpart* x_gpu;
    hipMalloc(&x_gpu, sizeof(FPpart)*part->npmax);

    FPpart* y_gpu;
    hipMalloc(&y_gpu, sizeof(FPpart)*part->npmax);

    FPpart* z_gpu;
    hipMalloc(&z_gpu, sizeof(FPpart)*part->npmax);

    FPpart* u_gpu;
    hipMalloc(&u_gpu, sizeof(FPpart)*part->npmax);

    FPpart* v_gpu;
    hipMalloc(&v_gpu, sizeof(FPpart)*part->npmax);

    FPpart* w_gpu;
    hipMalloc(&w_gpu, sizeof(FPpart)*part->npmax);

    FPinterp* q_gpu;
    hipMalloc(&q_gpu, sizeof(FPinterp)*part->npmax);

    // Copy array values to the device
    hipMemcpy(x_gpu, part->x, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);
    hipMemcpy(y_gpu, part->y, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);
    hipMemcpy(z_gpu, part->z, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);
    hipMemcpy(u_gpu, part->u, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);
    hipMemcpy(v_gpu, part->v, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);
    hipMemcpy(w_gpu, part->w, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);
    hipMemcpy(q_gpu, part->q, sizeof(FPinterp)*part->npmax, hipMemcpyHostToDevice);

    // Create temporary copy of host pointers
    FPpart* x_host = part->x;
    FPpart* y_host = part->y;
    FPpart* z_host = part->z;
    FPpart* u_host = part->u;
    FPpart* v_host = part->v;
    FPpart* w_host = part->w;
    FPinterp* q_host = part->q;

    // Point to device pointer in host struct
    part->x = x_gpu;
    part->y = y_gpu;
    part->z = z_gpu;
    part->u = u_gpu;
    part->v = v_gpu;
    part->w = w_gpu;
    part->q = q_gpu;

    // Move data to the GPU (pointers still pointing to host addresses)
    hipMemcpy(*part_gpu, part, sizeof(particles), hipMemcpyHostToDevice); 

    // Restore host pointer
    part->x = x_host;
    part->y = y_host;
    part->z = z_host;
    part->u = u_host;
    part->v = v_host;
    part->w = w_host;
    part->q = q_host; 
}

/** move particle array to CPU */
void particle_move2cpu(struct particles* part_gpu, struct particles* part)
{   
    // Create temporary copy of host pointers
    FPpart* x_host = part->x;
    FPpart* y_host = part->y;
    FPpart* z_host = part->z;
    FPpart* u_host = part->u;
    FPpart* v_host = part->v;
    FPpart* w_host = part->w;
    FPinterp* q_host = part->q;

    // Move data to the CPU
    hipMemcpy(part, part_gpu, sizeof(particles), hipMemcpyDeviceToHost);

    // Create temporary copy of device pointers
    FPpart* x_device = part->x;
    FPpart* y_device = part->y;
    FPpart* z_device = part->z;
    FPpart* u_device = part->u;
    FPpart* v_device = part->v;
    FPpart* w_device = part->w;
    FPinterp* q_device = part->q;

    // Restore host pointer
    part->x = x_host;
    part->y = y_host;
    part->z = z_host;
    part->u = u_host;
    part->v = v_host;
    part->w = w_host;
    part->q = q_host;
    
    // move particle arrays
    hipMemcpy(part->x, x_device, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->y, y_device, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->z, z_device, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->u, u_device, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->v, v_device, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->w, w_device, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->q, q_device, sizeof(FPinterp)*part->npmax, hipMemcpyDeviceToHost);
}

/** deallocate */
void particle_deallocate_gpu(struct particles* part_gpu)
{
    // Create temporary struct
    particles* part_tmp = new particles;
    hipMemcpy(part_tmp, part_gpu, sizeof(particles), hipMemcpyDeviceToHost);

    // deallocate particle variables
    hipFree(part_tmp->x);
    hipFree(part_tmp->y);
    hipFree(part_tmp->z);
    hipFree(part_tmp->u);
    hipFree(part_tmp->v);
    hipFree(part_tmp->w);
    hipFree(part_tmp->q);
    hipFree(part_gpu);
}


/** move EMfield to GPU */
void emfield_move2gpu(struct EMfield* field, struct EMfield** field_gpu, struct grid* grd)
{   
    // Allocate memory on the GPU
    hipMalloc(field_gpu, sizeof(EMfield));

    // Allocate arrays on the device
    FPfield* Ex_flat_gpu;
    hipMalloc(&Ex_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    FPfield* Ey_flat_gpu;
    hipMalloc(&Ey_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    FPfield* Ez_flat_gpu;
    hipMalloc(&Ez_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    FPfield* Bxn_flat_gpu;
    hipMalloc(&Bxn_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    FPfield* Byn_flat_gpu;
    hipMalloc(&Byn_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    FPfield* Bzn_flat_gpu;
    hipMalloc(&Bzn_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    // Copy array values to the device
    hipMemcpy(Ex_flat_gpu, field->Ex_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(Ey_flat_gpu, field->Ey_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(Ez_flat_gpu, field->Ez_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(Bxn_flat_gpu, field->Bxn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(Byn_flat_gpu, field->Byn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(Bzn_flat_gpu, field->Bzn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    // Create temporary copy of host pointers
    FPfield* Ex_flat_host = field->Ex_flat;
    FPfield* Ey_flat_host = field->Ey_flat;
    FPfield* Ez_flat_host = field->Ez_flat;
    FPfield* Bxn_flat_host = field->Bxn_flat;
    FPfield* Byn_flat_host = field->Byn_flat;
    FPfield* Bzn_flat_host = field->Bzn_flat;

    // Point to device pointer in host struct
    field->Ex_flat = Ex_flat_gpu;
    field->Ey_flat = Ey_flat_gpu;
    field->Ez_flat = Ez_flat_gpu;
    field->Bxn_flat = Bxn_flat_gpu;
    field->Byn_flat = Byn_flat_gpu;
    field->Bzn_flat = Bzn_flat_gpu;


    // Move data to the GPU (pointers still pointing to host addresses)
    hipMemcpy(*field_gpu, field, sizeof(EMfield), hipMemcpyHostToDevice); 

    // Restore host pointer
    field->Ex_flat = Ex_flat_host;
    field->Ey_flat = Ey_flat_host;
    field->Ez_flat = Ez_flat_host;
    field->Bxn_flat = Bxn_flat_host;
    field->Byn_flat = Byn_flat_host;
    field->Bzn_flat = Bzn_flat_host;   
}

/** move EMfield to CPU */
void emfield_move2cpu(struct EMfield* field_gpu, struct EMfield* field, struct grid* grd)
{    
    // Create temporary copy of host pointers
    FPfield* Ex_flat_host = field->Ex_flat;
    FPfield* Ey_flat_host = field->Ey_flat;
    FPfield* Ez_flat_host = field->Ez_flat;
    FPfield* Bxn_flat_host = field->Bxn_flat;
    FPfield* Byn_flat_host = field->Byn_flat;
    FPfield* Bzn_flat_host = field->Bzn_flat;

    // Move data to the CPU
    hipMemcpy(field, field_gpu, sizeof(EMfield), hipMemcpyDeviceToHost);

    // Create temporary copy of device pointers
    FPfield* Ex_flat_device = field->Ex_flat;
    FPfield* Ey_flat_device = field->Ey_flat;
    FPfield* Ez_flat_device = field->Ez_flat;
    FPfield* Bxn_flat_device = field->Bxn_flat;
    FPfield* Byn_flat_device = field->Byn_flat;
    FPfield* Bzn_flat_device = field->Bzn_flat;

    // Restore host pointer
    field->Ex_flat = Ex_flat_host;
    field->Ey_flat = Ey_flat_host;
    field->Ez_flat = Ez_flat_host;
    field->Bxn_flat = Bxn_flat_host;
    field->Byn_flat = Byn_flat_host;
    field->Bzn_flat = Bzn_flat_host;

    // move arrays
    hipMemcpy(field->Ex_flat, Ex_flat_device, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Ey_flat, Ey_flat_device, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Ez_flat, Ez_flat_device, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Bxn_flat, Bxn_flat_device, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Byn_flat, Byn_flat_device, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Bzn_flat, Bzn_flat_device, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);  
}

/** deallocate */
void emfield_deallocate_gpu(struct EMfield* field_gpu)
{
    // Create temporary struct
    EMfield* field_tmp = new EMfield;
    hipMemcpy(field_tmp, field_gpu, sizeof(EMfield), hipMemcpyDeviceToHost);

    // deallocate variables
    hipFree(field_tmp->Ex_flat);
    hipFree(field_tmp->Ey_flat);
    hipFree(field_tmp->Ez_flat);
    hipFree(field_tmp->Bxn_flat);
    hipFree(field_tmp->Byn_flat);
    hipFree(field_tmp->Bzn_flat);
    hipFree(field_gpu);
}


/** move grid to GPU */
void grid_move2gpu(struct grid* grd, struct grid** grd_gpu)
{   
    // Allocate memory on the GPU
    hipMalloc(grd_gpu, sizeof(grid)); 

    // Allocate arrays on the device
    FPfield* XN_flat_gpu;
    hipMalloc(&XN_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    FPfield* YN_flat_gpu;
    hipMalloc(&YN_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    
    FPfield* ZN_flat_gpu;
    hipMalloc(&ZN_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    // Copy array values to the device
    hipMemcpy(XN_flat_gpu, grd->XN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(YN_flat_gpu, grd->YN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(ZN_flat_gpu, grd->ZN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    // Create temporary copy of host pointers
    FPfield* XN_flat_host = grd->XN_flat;
    FPfield* YN_flat_host = grd->YN_flat;
    FPfield* ZN_flat_host = grd->ZN_flat;

    // Point to device pointer in host struct
    grd->XN_flat = XN_flat_gpu;
    grd->YN_flat = YN_flat_gpu;
    grd->ZN_flat = ZN_flat_gpu;


    // Move data to the GPU (pointers still pointing to host addresses)
    hipMemcpy(*grd_gpu, grd, sizeof(grid), hipMemcpyHostToDevice); 

    // Restore host pointer
    grd->XN_flat = XN_flat_host;
    grd->YN_flat = YN_flat_host;
    grd->ZN_flat = ZN_flat_host;
}

/** move grid to CPU */
void grid_move2cpu(struct grid* grd_gpu, struct grid* grd)
{    
    // Create temporary copy of host pointers
    FPfield* XN_flat_host = grd->XN_flat;
    FPfield* YN_flat_host = grd->YN_flat;
    FPfield* ZN_flat_host = grd->ZN_flat;

    // Move data to the CPU
    hipMemcpy(grd, grd_gpu, sizeof(grid), hipMemcpyDeviceToHost);

    // Create temporary copy of device pointers
    FPfield* XN_flat_device = grd->XN_flat;
    FPfield* YN_flat_device = grd->YN_flat;
    FPfield* ZN_flat_device = grd->ZN_flat;

    // Restore host pointer
    grd->XN_flat = XN_flat_host;
    grd->YN_flat = YN_flat_host;
    grd->ZN_flat = ZN_flat_host;

    // move arrays
    hipMemcpy(grd->XN_flat, XN_flat_device, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(grd->YN_flat, YN_flat_device, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(grd->ZN_flat, ZN_flat_device, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);
}

/** deallocate */
void grid_deallocate_gpu(struct grid* grd_gpu)
{
    // Create temporary struct
    grid* grd_tmp = new grid;
    hipMemcpy(grd_tmp, grd_gpu, sizeof(grid), hipMemcpyDeviceToHost);

    // deallocate variables
    hipFree(grd_tmp->XN_flat);
    hipFree(grd_tmp->YN_flat);
    hipFree(grd_tmp->ZN_flat);

    hipFree(grd_gpu);
}
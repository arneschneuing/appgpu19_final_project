#include "GPUAllocation.h"

/** move particle array to GPU */
void particle_move2gpu(struct particles* part, struct particles* part_gpu)
{   
    // Allocate memory on the GPU
    hipMalloc(&part_gpu, sizeof(particles)); 

    // Allocate arrays on the device
    FPpart* x_gpu;
    hipMalloc(&x_gpu, sizeof(FPpart)*part->npmax);

    FPpart* y_gpu;
    hipMalloc(&y_gpu, sizeof(FPpart)*part->npmax);

    FPpart* z_gpu;
    hipMalloc(&z_gpu, sizeof(FPpart)*part->npmax);

    FPpart* u_gpu;
    hipMalloc(&u_gpu, sizeof(FPpart)*part->npmax);

    FPpart* v_gpu;
    hipMalloc(&v_gpu, sizeof(FPpart)*part->npmax);

    FPpart* w_gpu;
    hipMalloc(&w_gpu, sizeof(FPpart)*part->npmax);

    FPinterp* q_gpu;
    hipMalloc(&q_gpu, sizeof(FPinterp)*part->npmax);

    // Copy array values to the device
    hipMemcpy(x_gpu, part->x, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);
    hipMemcpy(y_gpu, part->y, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);
    hipMemcpy(z_gpu, part->z, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);
    hipMemcpy(u_gpu, part->u, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);
    hipMemcpy(v_gpu, part->v, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);
    hipMemcpy(w_gpu, part->w, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);
    hipMemcpy(q_gpu, part->q, sizeof(FPinterp)*part->npmax, hipMemcpyHostToDevice);



    // Create temporary copy of host pointers
    FPpart* x_host = part->x;
    FPpart* y_host = part->y;
    FPpart* z_host = part->z;
    FPpart* u_host = part->u;
    FPpart* v_host = part->v;
    FPpart* w_host = part->w;
    FPinterp* q_host = part->q;

    // Point to device pointer in host struct
    part->x = x_gpu;
    part->y = y_gpu;
    part->z = z_gpu;
    part->u = u_gpu;
    part->v = v_gpu;
    part->w = w_gpu;
    part->q = q_gpu;

    // Move data to the GPU (pointers still pointing to host addresses)
    hipMemcpy(part_gpu, part, sizeof(particles), hipMemcpyHostToDevice); 

    // Restore host pointer
    part->x = x_host;
    part->y = y_host;
    part->z = z_host;
    part->u = u_host;
    part->v = v_host;
    part->w = w_host;
    part->q = q_host; 
}

/** move particle array to CPU */
void particle_move2cpu(struct particles* part_gpu, struct particles* part)
{   
    // Move data to the CPU
    hipMemcpy(part, part_gpu, sizeof(particles), hipMemcpyDeviceToHost);    
    
    // move particle arrays
    hipMemcpy(part->x, part_gpu->x, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->y, part_gpu->y, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->z, part_gpu->z, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->u, part_gpu->u, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->v, part_gpu->v, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->w, part_gpu->w, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->q, part_gpu->q, sizeof(FPinterp)*part->npmax, hipMemcpyDeviceToHost);    
}

/** deallocate */
void particle_deallocate_gpu(struct particles* part_gpu)
{
    // deallocate particle variables
    hipFree(part_gpu->x);
    hipFree(part_gpu->y);
    hipFree(part_gpu->z);
    hipFree(part_gpu->u);
    hipFree(part_gpu->v);
    hipFree(part_gpu->w);
    hipFree(part_gpu->q);
    hipFree(part_gpu);
}


/** move EMfield to GPU */
void emfield_move2gpu(struct EMfield* field, struct EMfield* field_gpu, struct grid* grd)
{   
    // Allocate memory on the GPU
    hipMalloc(&field_gpu, sizeof(EMfield));

    // Allocate arrays on the device
    FPfield* Ex_flat_gpu;
    hipMalloc(&Ex_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    FPfield* Ey_flat_gpu;
    hipMalloc(&Ey_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    FPfield* Ez_flat_gpu;
    hipMalloc(&Ez_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    FPfield* Bxn_flat_gpu;
    hipMalloc(&Bxn_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    FPfield* Byn_flat_gpu;
    hipMalloc(&Byn_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    FPfield* Bzn_flat_gpu;
    hipMalloc(&Bzn_flat_gpu, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    // Copy array values to the device
    hipMemcpy(Ex_flat_gpu, field->Ex_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(Ey_flat_gpu, field->Ey_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(Ez_flat_gpu, field->Ez_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(Bxn_flat_gpu, field->Bxn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(Byn_flat_gpu, field->Byn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(Bzn_flat_gpu, field->Bzn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    // Create temporary copy of host pointers
    FPfield* Ex_flat_host = field->Ex_flat;
    FPfield* Ey_flat_host = field->Ey_flat;
    FPfield* Ez_flat_host = field->Ez_flat;
    FPfield* Bxn_flat_host = field->Bxn_flat;
    FPfield* Byn_flat_host = field->Byn_flat;
    FPfield* Bzn_flat_host = field->Bzn_flat;

    // Point to device pointer in host struct
    field->Ex_flat = Ex_flat_gpu;
    field->Ey_flat = Ey_flat_gpu;
    field->Ez_flat = Ez_flat_gpu;
    field->Bxn_flat = Bxn_flat_gpu;
    field->Byn_flat = Byn_flat_gpu;
    field->Bzn_flat = Bzn_flat_gpu;


    // Move data to the GPU (pointers still pointing to host addresses)
    hipMemcpy(field_gpu, field, sizeof(EMfield), hipMemcpyHostToDevice); 

    // Restore host pointer
    field->Ex_flat = Ex_flat_host;
    field->Ey_flat = Ey_flat_host;
    field->Ez_flat = Ez_flat_host;
    field->Bxn_flat = Bxn_flat_host;
    field->Byn_flat = Byn_flat_host;
    field->Bzn_flat = Bzn_flat_host;   
}

/** move EMfield to CPU */
void emfield_move2cpu(struct EMfield* field_gpu, struct EMfield* field, struct grid* grd)
{    
    // move arrays
    hipMemcpy(field->Ex_flat, field_gpu->Ex_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Ey_flat, field_gpu->Ey_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Ez_flat, field_gpu->Ez_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Bxn_flat, field_gpu->Bxn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Byn_flat, field_gpu->Byn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Bzn_flat, field_gpu->Bzn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);  
}

/** deallocate */
void emfield_deallocate_gpu(struct EMfield* field_gpu)
{
    // deallocate variables
    hipFree(field_gpu->Ex_flat);
    hipFree(field_gpu->Ey_flat);
    hipFree(field_gpu->Ez_flat);
    hipFree(field_gpu->Bxn_flat);
    hipFree(field_gpu->Byn_flat);
    hipFree(field_gpu->Bzn_flat);
    hipFree(field_gpu);
}


/** move grid to GPU */
void grid_move2gpu(struct grid* grd, struct grid* grd_gpu)
{   
    // Allocate memory on the GPU
    hipMalloc(&grd_gpu, sizeof(particles)); 

    // Allocate arrays on the device
    FPfield* XN_flat_gpu;
    hipMalloc(&XN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    FPfield* YN_flat_gpu;
    hipMalloc(&YN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    
    FPfield* ZN_flat_gpu;
    hipMalloc(&ZN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    // Copy array values to the device
    hipMemcpy(XN_flat_gpu, grd->XN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(YN_flat_gpu, grd->YN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(ZN_flat_gpu, grd->ZN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    // Create temporary copy of host pointers
    FPfield* XN_flat_host = grd->ZN;
    FPfield* YN_flat_host = grd->YN;
    FPfield* ZN_flat_host = grd->ZN;

    // Point to device pointer in host struct
    grd->XN_flat = XN_flat_gpu;
    grd->YN_flat = YN_flat_gpu;
    grd->ZN_flat = ZN_flat_gpu;


    // Move data to the GPU (pointers still pointing to host addresses)
    hipMemcpy(grd_gpu, grd, sizeof(grid), hipMemcpyHostToDevice); 

    // Restore host pointer
    grid->XN_flat = XN_flat_host;
    grid->YN_flat = YN_flat_host;
    grid->ZN_flat = ZN_flat_host;
}

/** move grid to CPU */
void grid_move2cpu(struct grid* grd_gpu, struct grid* grd)
{    
    // move arrays
    hipMemcpy(grd->XN_flat, grd_gpu->XN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(grd->YN_flat, grd_gpu->YN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(grd->ZN_flat, grd_gpu->ZN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);
}

/** deallocate */
void grid_deallocate_gpu(struct grid* grd_gpu)
{
    // deallocate variables
    hipFree(grd_gpu->XN_flat);
    hipFree(grd_gpu->YN_flat);
    hipFree(grd_gpu->ZN_flat);

    hipFree(grd_gpu);
}
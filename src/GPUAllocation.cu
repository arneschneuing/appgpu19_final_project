#include "GPUAllocation.h"

/** move particle array to GPU */
void particle_move2gpu(struct particles* part, struct particles* part_gpu)
{   
    // Allocate memory on the GPU
    hipMalloc(&part_gpu, sizeof(particles)); 

    // Allocate arrays on the device
    FPfield* x_gpu;
    hipMalloc(&x_gpu, sizeof(FPpart)*part->npmax);

    // Copy array values to the device
    hipMemcpy(x_gpu, part->x, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);

    // Create temporary copy of host pointers
    FPfield* x_host = part->x;

    // Point to device pointer in host struct
    part->x = x_gpu;

    // Move data to the GPU (pointers still pointing to host addresses)
    hipMemcpy(part_gpu, part, sizeof(particles), hipMemcpyHostToDevice); 

    // Restore host pointer
    part->x = x_host;

    std::cout << "Hello" << std::endl;
    
    // move particle arrays
    hipMalloc(&part_gpu->x, sizeof(FPpart)*part->npmax);
    std::cout << "Hello" << std::endl;
    hipMemcpy(part_gpu->x, part->x, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);
    std::cout << "Hello" << std::endl;

    hipMalloc(&part_gpu->y, sizeof(FPpart)*part->npmax);
    hipMemcpy(part_gpu->y, part->y, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);

    hipMalloc(&part_gpu->z, sizeof(FPpart)*part->npmax);
    hipMemcpy(part_gpu->z, part->z, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);

    hipMalloc(&part_gpu->u, sizeof(FPpart)*part->npmax);
    hipMemcpy(part_gpu->u, part->u, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);

    hipMalloc(&part_gpu->v, sizeof(FPpart)*part->npmax);
    hipMemcpy(part_gpu->v, part->v, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);

    hipMalloc(&part_gpu->w, sizeof(FPpart)*part->npmax);
    hipMemcpy(part_gpu->w, part->w, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);

    hipMalloc(&part_gpu->q, sizeof(FPinterp)*part->npmax);
    hipMemcpy(part_gpu->q, part->q, sizeof(FPinterp)*part->npmax, hipMemcpyHostToDevice);    
}

/** move particle array to CPU */
void particle_move2cpu(struct particles* part_gpu, struct particles* part)
{   
    // Move data to the CPU
    hipMemcpy(part, part_gpu, sizeof(particles), hipMemcpyDeviceToHost);    
    
    // move particle arrays
    hipMemcpy(part->x, part_gpu->x, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->y, part_gpu->y, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->z, part_gpu->z, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->u, part_gpu->u, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->v, part_gpu->v, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->w, part_gpu->w, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->q, part_gpu->q, sizeof(FPinterp)*part->npmax, hipMemcpyDeviceToHost);    
}

/** deallocate */
void particle_deallocate_gpu(struct particles* part_gpu)
{
    // deallocate particle variables
    hipFree(part_gpu->x);
    hipFree(part_gpu->y);
    hipFree(part_gpu->z);
    hipFree(part_gpu->u);
    hipFree(part_gpu->v);
    hipFree(part_gpu->w);
    hipFree(part_gpu->q);
    hipFree(part_gpu);
}


/** move EMfield to GPU */
void emfield_move2gpu(struct EMfield* field, struct EMfield* field_gpu, struct grid* grd)
{   
    // Allocate memory on the GPU
    hipMalloc(&field_gpu, sizeof(EMfield));   
    
    // Copy arrays
    hipMalloc(&field_gpu->Ex_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    hipMemcpy(field_gpu->Ex_flat, field->Ex_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    hipMalloc(&field_gpu->Ey_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    hipMemcpy(field_gpu->Ey_flat, field->Ey_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    hipMalloc(&field_gpu->Ez_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    hipMemcpy(field_gpu->Ez_flat, field->Ez_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    hipMalloc(&field_gpu->Bxn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    hipMemcpy(field_gpu->Bxn_flat, field->Bxn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    hipMalloc(&field_gpu->Byn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    hipMemcpy(field_gpu->Byn_flat, field->Byn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    hipMalloc(&field_gpu->Bzn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    hipMemcpy(field_gpu->Bzn_flat, field->Bzn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
   
}

/** move EMfield to CPU */
void emfield_move2cpu(struct EMfield* field_gpu, struct EMfield* field, struct grid* grd)
{    
    // move arrays
    hipMemcpy(field->Ex_flat, field_gpu->Ex_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Ey_flat, field_gpu->Ey_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Ez_flat, field_gpu->Ez_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Bxn_flat, field_gpu->Bxn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Byn_flat, field_gpu->Byn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Bzn_flat, field_gpu->Bzn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);  
}

/** deallocate */
void emfield_deallocate_gpu(struct EMfield* field_gpu)
{
    // deallocate variables
    hipFree(field_gpu->Ex_flat);
    hipFree(field_gpu->Ey_flat);
    hipFree(field_gpu->Ez_flat);
    hipFree(field_gpu->Bxn_flat);
    hipFree(field_gpu->Byn_flat);
    hipFree(field_gpu->Bzn_flat);
    hipFree(field_gpu);
}


/** move grid to GPU */
void grid_move2gpu(struct grid* grd, struct grid* grd_gpu)
{   
    // Allocate memory on the GPU
    hipMalloc(&grd_gpu, sizeof(particles)); 

    // Allocate arrays on the device
    FPfield* XN_flat_gpu;
    hipMalloc(&XN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    FPfield* YN_flat_gpu;
    hipMalloc(&YN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    
    FPfield* ZN_flat_gpu;
    hipMalloc(&ZN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);

    // Copy array values to the device
    hipMemcpy(XN_flat_gpu, grd->XN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(YN_flat_gpu, grd->YN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
    hipMemcpy(ZN_flat_gpu, grd->ZN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    // Create temporary copy of host pointers
    FPfield* XN_flat_host = grd->ZN;
    FPfield* YN_flat_host = grd->YN;
    FPfield* ZN_flat_host = grd->ZN;

    // Point to device pointer in host struct
    grd->XN_flat = XN_flat_gpu;
    grd->YN_flat = YN_flat_gpu;
    grd->ZN_flat = ZN_flat_gpu;


    // Move data to the GPU (pointers still pointing to host addresses)
    hipMemcpy(grd_gpu, grd, sizeof(grid), hipMemcpyHostToDevice); 

    // Restore host pointer
    grid->XN_flat = XN_flat_host;
    grid->YN_flat = YN_flat_host;
    grid->ZN_flat = ZN_flat_host;
}

/** move grid to CPU */
void grid_move2cpu(struct grid* grd_gpu, struct grid* grd)
{    
    // move arrays
    hipMemcpy(grd->XN_flat, grd_gpu->XN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(grd->YN_flat, grd_gpu->YN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(grd->ZN_flat, grd_gpu->ZN_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);
}

/** deallocate */
void grid_deallocate_gpu(struct grid* grd_gpu)
{
    // deallocate variables
    hipFree(grd_gpu->XN_flat);
    hipFree(grd_gpu->YN_flat);
    hipFree(grd_gpu->ZN_flat);

    hipFree(grd_gpu);
}
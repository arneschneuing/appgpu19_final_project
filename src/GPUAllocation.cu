#include "Particles.h"

/** move particle array to GPU */
void particle_move2gpu(struct particles* part, struct particles* part_gpu)
{   
    // Allocate memory on the GPU
    hipMalloc(&part_gpu, sizeof(particles));

    // Move data to the GPU
    hipMemcpy(part_gpu, part, sizeof(particles), hipMemcpyHostToDevice);    
    
    // move particle arrays
    hipMalloc(&part_gpu->x, sizeof(FPpart)*part->npmax);
    hipMemcpy(part_gpu->x, part->x, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);

    hipMalloc(&part_gpu->y, sizeof(FPpart)*part->npmax);
    hipMemcpy(part_gpu->y, part->y, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);

    hipMalloc(&part_gpu->z, sizeof(FPpart)*part->npmax);
    hipMemcpy(part_gpu->z, part->z, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);

    hipMalloc(&part_gpu->u, sizeof(FPpart)*part->npmax);
    hipMemcpy(part_gpu->u, part->u, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);

    hipMalloc(&part_gpu->v, sizeof(FPpart)*part->npmax);
    hipMemcpy(part_gpu->v, part->v, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);

    hipMalloc(&part_gpu->w, sizeof(FPpart)*part->npmax);
    hipMemcpy(part_gpu->w, part->w, sizeof(FPpart)*part->npmax, hipMemcpyHostToDevice);

    hipMalloc(&part_gpu->q, sizeof(FPinterp)*part->npmax);
    hipMemcpy(part_gpu->q, part->q, sizeof(FPinterp)*part->npmax, hipMemcpyHostToDevice);    
}

/** move particle array to CPU */
void particle_move2cpu(struct particles* part_gpu, struct particles* part)
{   
    // Move data to the CPU
    hipMemcpy(part, part_gpu, sizeof(particles), hipMemcpyDeviceToHost);    
    
    // move particle arrays
    hipMemcpy(part->x, part_gpu->x, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->y, part_gpu->y, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->z, part_gpu->z, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->u, part_gpu->u, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->v, part_gpu->v, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->w, part_gpu->w, sizeof(FPpart)*part->npmax, hipMemcpyDeviceToHost);

    hipMemcpy(part->q, part_gpu->q, sizeof(FPinterp)*part->npmax, hipMemcpyDeviceToHost);    
}

/** deallocate */
void particle_deallocate_gpu(struct particles* part_gpu)
{
    // deallocate particle variables
    hipFree(part_gpu->x);
    hipFree(part_gpu->y);
    hipFree(part_gpu->z);
    hipFree(part_gpu->u);
    hipFree(part_gpu->v);
    hipFree(part_gpu->w);
    hipFree(part_gpu->q);
    hipFree(part_gpu);
}


/** move EMfield to GPU */
void emfield_move2gpu(struct EMfield* field, struct EMfield* field_gpu, struct grid* grd)
{   
    // Allocate memory on the GPU
    hipMalloc(&field_gpu, sizeof(EMfield));   
    
    // Copy arrays
    hipMalloc(&field_gpu->Ex_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    hipMemcpy(field_gpu->Ex_flat, field->Ex_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    hipMalloc(&field_gpu->Ey_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    hipMemcpy(field_gpu->Ey_flat, field->Ey_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    hipMalloc(&field_gpu->Ez_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    hipMemcpy(field_gpu->Ez_flat, field->Ez_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    hipMalloc(&field_gpu->Bxn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    hipMemcpy(field_gpu->Bxn_flat, field->Bxn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    hipMalloc(&field_gpu->Byn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    hipMemcpy(field_gpu->Byn_flat, field->Byn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);

    hipMalloc(&field_gpu->Bzn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn);
    hipMemcpy(field_gpu->Bzn_flat, field->Bzn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyHostToDevice);
   
}

/** move EMfield to CPU */
void emfield_move2cpu(struct EMfield* field_gpu, struct EMfield* field, struct grid* grd)
{    
    // move arrays
    hipMemcpy(field->Ex_flat, field_gpu->Ex_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Ey_flat, field_gpu->Ey_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Ez_flat, field_gpu->Ez_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Bxn_flat, field_gpu->Bxn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Byn_flat, field_gpu->Byn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);

    hipMemcpy(field->Bzn_flat, field_gpu->Bzn_flat, sizeof(FPfield) * grd->nxn * grd->nyn * grd->nzn, hipMemcpyDeviceToHost);  
}

/** deallocate */
void emfield_deallocate_gpu(struct EMfield* field_gpu)
{
    // deallocate variables
    hipFree(field_gpu->Ex);
    hipFree(field_gpu->Ey);
    hipFree(field_gpu->Ez);
    hipFree(field_gpu->Bxn);
    hipFree(field_gpu->Byn);
    hipFree(field_gpu->Bzn);
    hipFree(field_gpu);
}